#include "core/builder.h"
#include "gtest/gtest.h"
#include "test_utils.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

namespace arboretum_test {

TEST(SingleNodeHistSumFloat, Naive) {
  const size_t size = 1 << 5;
  thrust::device_vector<float> grad(size);
  thrust::device_vector<float> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = i;
  }

  arboretum::core::HistTreeGrower<unsigned, float, float>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 6, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  for (unsigned i = 0; i < size; ++i) {
    ASSERT_EQ(count[i], 1);
    ASSERT_FLOAT_EQ(grad[i], sum[i]);
  }
}

TEST(SingleNodeHistSumFloat, SingleSegment) {
  const size_t size = 1 << 5;
  thrust::device_vector<float> grad(size);
  thrust::device_vector<float> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0;
  }

  arboretum::core::HistTreeGrower<unsigned, float, float>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 6, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size);
  // sum of 0 + 1 + .. + size-1
  float true_sum = size * (size - 1) / 2;
  ASSERT_FLOAT_EQ(sum[0], float(true_sum));
  //   }
}

TEST(SingleNodeHistSumFloat, SingleSegmentFullSize) {
  const size_t size = HIST_SUM_BLOCK_DIM * HIST_SUM_ITEMS_PER_THREAD;

  thrust::device_vector<float> grad(size);
  thrust::device_vector<float> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0 % 1024;
  }

  arboretum::core::HistTreeGrower<unsigned, float, float>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 11, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size);
  // sum of 0 + 1 + .. + size-1
  float true_sum = size * (size - 1) / 2;
  ASSERT_FLOAT_EQ(sum[0], float(true_sum));
  //   }
}

TEST(SingleNodeHistSumDouble, Naive) {
  const size_t size = 1 << 5;
  thrust::device_vector<float> grad(size);
  thrust::device_vector<double> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = i;
  }

  arboretum::core::HistTreeGrower<unsigned, float, double>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 6, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  for (unsigned i = 0; i < size; ++i) {
    ASSERT_EQ(count[i], 1);
    ASSERT_DOUBLE_EQ(grad[i], sum[i]);
  }
}

TEST(SingleNodeHistSumDouble, SingleSegment) {
  const size_t size = 1 << 5;
  thrust::device_vector<float> grad(size);
  thrust::device_vector<double> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0;
  }

  arboretum::core::HistTreeGrower<unsigned, float, double>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 6, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size);
  // sum of 0 + 1 + .. + size-1
  double true_sum = size * (size - 1) / 2;
  ASSERT_DOUBLE_EQ(sum[0], true_sum);
  //   }
}

TEST(SingleNodeHistSumDouble, SingleSegmentFullSize) {
  const size_t size = HIST_SUM_BLOCK_DIM * HIST_SUM_ITEMS_PER_THREAD;

  thrust::device_vector<float> grad(size);
  thrust::device_vector<double> sum(size, 0.0);
  thrust::device_vector<unsigned> count(size, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(2);
  node_size[0] = 0;
  node_size[1] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0 % 1024;
  }

  arboretum::core::HistTreeGrower<unsigned, float, double>::HistSumSingleNode(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 11, size);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size);
  // sum of 0 + 1 + .. + size-1
  double true_sum = size * (size - 1) / 2;
  ASSERT_DOUBLE_EQ(sum[0], true_sum);
  //   }
}

TEST(MultiNodeHistSumDouble, SingleSegmentNoTrick) {
  const unsigned hist_size = 4;
  const size_t size = HIST_SUM_BLOCK_DIM * HIST_SUM_ITEMS_PER_THREAD;

  thrust::device_vector<float> grad(size);
  thrust::device_vector<double> sum(hist_size * 2, 0.0);
  thrust::device_vector<unsigned> count(hist_size * 2, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(3);
  node_size[0] = 0;
  node_size[1] = size / 2;
  node_size[2] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0;
  }

  // SUM_T *sum, unsigned *bin_count,
  //                       const SUM_T *hist_sum_parent,
  //                       const unsigned *hist_count_parent, const GRAD_T
  //                       *grad, const unsigned *node_size, const unsigned
  //                       *fvalue, const unsigned hist_size_bits, const
  //                       unsigned hist_size, const unsigned size, const bool
  //                       use_trick, hipStream_t stream = 0

  arboretum::core::HistTreeGrower<unsigned, float, double>::HistSum(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()), NULL, NULL,
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 10, hist_size, 2, false);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size / 2);
  ASSERT_EQ(count[hist_size], size / 2);
  // sum of 0 + 1 + .. + size / 2 -1
  double true_sum = (size / 2) * (size / 2 - 1) / 2;
  ASSERT_DOUBLE_EQ(sum[0], true_sum);

  // sum of size / 2 + ... + size -1

  true_sum = (size / 2) * (size / 2 + size - 1) / 2;
  ASSERT_DOUBLE_EQ(sum[hist_size], true_sum);
  //   }
}

TEST(MultiNodeHistSumDouble, SingleSegmentWithTrick) {
  const unsigned hist_size = 4;
  const size_t size = HIST_SUM_BLOCK_DIM * HIST_SUM_ITEMS_PER_THREAD;

  thrust::device_vector<float> grad(size);
  thrust::device_vector<double> sum(hist_size * 2, 0.0);
  thrust::device_vector<unsigned> count(hist_size * 2, 0);
  thrust::device_vector<unsigned> bin(size);
  thrust::device_vector<unsigned> node_size(3);
  node_size[0] = 0;
  node_size[1] = size / 2;
  node_size[2] = size;
  for (unsigned i = 0; i < size; ++i) {
    grad[i] = float(i);
    bin[i] = 0;
  }

  thrust::device_vector<unsigned> parent_count(hist_size, 0);
  thrust::device_vector<double> parent_sum(hist_size, 0);
  parent_count[0] = size;
  // sum of 0 + 1 + .. + size / 2 -1
  double true_sum = (size) * (size - 1) / 2;
  parent_sum[0] = true_sum;

  // SUM_T *sum, unsigned *bin_count,
  //                       const SUM_T *hist_sum_parent,
  //                       const unsigned *hist_count_parent, const GRAD_T
  //                       *grad, const unsigned *node_size, const unsigned
  //                       *fvalue, const unsigned hist_size_bits, const
  //                       unsigned hist_size, const unsigned size, const bool
  //                       use_trick, hipStream_t stream = 0

  arboretum::core::HistTreeGrower<unsigned, float, double>::HistSum(
    thrust::raw_pointer_cast(sum.data()),
    thrust::raw_pointer_cast(count.data()),
    thrust::raw_pointer_cast(parent_sum.data()),
    thrust::raw_pointer_cast(parent_count.data()),
    thrust::raw_pointer_cast(grad.data()),
    thrust::raw_pointer_cast(node_size.data()),
    thrust::raw_pointer_cast(bin.data()), 10, hist_size, 2, true);

  TEST_OK(hipDeviceSynchronize());
  TEST_OK(hipGetLastError());

  ASSERT_EQ(count[0], size / 2);
  ASSERT_EQ(count[hist_size], size / 2);
  // sum of 0 + 1 + .. + size / 2 -1
  true_sum = (size / 2) * (size / 2 - 1) / 2;

  ASSERT_DOUBLE_EQ(sum[0], true_sum);

  // sum of size / 2 + ... + size -1

  true_sum = (size / 2) * (size / 2 + size - 1) / 2;
  ASSERT_DOUBLE_EQ(sum[hist_size], true_sum);
  //   }
}

}  // namespace arboretum_test