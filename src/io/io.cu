#include "hip/hip_runtime.h"
//#include <omp.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/unique.h>
#include <algorithm>
#include <functional>
#include <unordered_set>
#include <vector>
#include "core/cuda_helpers.h"
#include "hipcub/hipcub.hpp"
#include "io.h"

namespace arboretum {
namespace io {
using namespace std;

#define ITEMS 8

template <int ITEMS_PER_THREAD>
__global__ void build_histogram(unsigned short *bin, float *threshold,
                                const float *fvalue_unique, const float *fvalue,
                                const int hist_size, const int unique_size,
                                const size_t n) {
  extern __shared__ float values[];
  const int size = min(unique_size, hist_size);
  if (threadIdx.x < hist_size) {
    values[threadIdx.x] = INFINITY;
    unsigned idx = (threadIdx.x + 1) * unique_size / size;
    if (threadIdx.x < size - 1)
      values[threadIdx.x] = (fvalue_unique[idx] + fvalue_unique[idx - 1]) * 0.5;
  }

  __syncthreads();

#pragma unroll
  for (unsigned i = 0; i < ITEMS_PER_THREAD; ++i) {
    unsigned idx =
      blockDim.x * blockIdx.x * ITEMS_PER_THREAD + i * blockDim.x + threadIdx.x;
    if (idx < n) bin[idx] = lower_bound<float>(values, fvalue[idx], size);
  }

  if (blockIdx.x == 0 && threadIdx.x < size)
    threshold[threadIdx.x] = values[threadIdx.x];
}

DataMatrix::DataMatrix(int rows, int columns, int columns_category)
    : rows(rows),
      columns(columns + columns_category),
      columns_dense(columns),
      columns_category(columns_category) {
  _init = false;
  data.resize(columns);
  data_category_device.resize(columns_category);
  sorted_data_device.resize(columns);
  data_reduced.resize(
    columns,
    thrust::host_vector<
      unsigned short,
      thrust::cuda::experimental::pinned_allocator<unsigned short>>(rows));
  reduced_size.resize(columns);
  category_size.resize(columns_category);
  data_reduced_mapping.resize(columns);
  data_categories.resize(columns_category);

  for (int i = 0; i < columns; ++i) {
    data[i].resize(rows);
  }
  for (int i = 0; i < columns_category; ++i) {
    data_categories[i].resize(rows);
  }
}

void DataMatrix::InitHist(int hist_size, bool verbose) {
  if (!_init) {
    thrust::host_vector<thrust::host_vector<float>> thresholds(columns_dense);
    thrust::device_vector<float> d_data(rows);
    thrust::device_vector<float> d_data_sorted(rows);
    thrust::device_vector<unsigned short> bin(rows);
    thrust::device_vector<float> d_threshold(hist_size);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes,
      thrust::raw_pointer_cast(d_data.data()),
      thrust::raw_pointer_cast(d_data_sorted.data()), rows);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    for (size_t i = 0; i < columns_dense; ++i) {
      thrust::copy(data[i].begin(), data[i].end(), d_data.begin());

      hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes,
        thrust::raw_pointer_cast(d_data.data()),
        thrust::raw_pointer_cast(d_data_sorted.data()), rows);

      auto n = thrust::unique(d_data_sorted.begin(), d_data_sorted.end());
      int unique_size = n - d_data_sorted.begin();

      int size = std::min(unique_size, hist_size);
      reduced_size[i] = 32 - __builtin_clz(size);
      data_reduced_mapping[i].resize(size);
      int grid_size = (rows + 1024 * ITEMS - 1) / (1024 * ITEMS);
      build_histogram<ITEMS><<<grid_size, 1024, hist_size * sizeof(float)>>>(
        thrust::raw_pointer_cast(bin.data()),
        thrust::raw_pointer_cast(d_threshold.data()),
        thrust::raw_pointer_cast(d_data_sorted.data()),
        thrust::raw_pointer_cast(d_data.data()), hist_size, unique_size, rows);

      OK(hipDeviceSynchronize());

      thrust::copy(d_threshold.begin(), d_threshold.begin() + size,
                   data_reduced_mapping[i].begin());

      //   data_reduced[i].resize(rows);

      thrust::copy(bin.begin(), bin.end(), data_reduced[i].begin());
    }

    OK(hipFree(d_temp_storage));

    for (size_t i = 0; i < columns_dense && verbose; ++i) {
      printf("feature %lu has been reduced to %u bits \n", i, reduced_size[i]);
    }
    max_reduced_size = max_feature_size =
      *std::max_element(reduced_size.begin(), reduced_size.end());
    if (verbose) printf("max feature size %u \n", max_reduced_size);

    this->_init = true;
  }
}

void DataMatrix::InitExact(bool verbose) {
  if (!_init) {
#pragma omp parallel for
    for (size_t i = 0; i < columns_dense; ++i) {
      data_reduced[i].resize(rows);

      std::unordered_set<float> s;
      for (float v : data[i]) s.insert(v);
      data_reduced_mapping[i].assign(s.begin(), s.end());
      std::sort(data_reduced_mapping[i].begin(), data_reduced_mapping[i].end());
      reduced_size[i] = 32 - __builtin_clz(data_reduced_mapping[i].size());

      for (size_t j = 0; j < rows; ++j) {
        vector<float>::iterator indx =
          std::lower_bound(data_reduced_mapping[i].begin(),
                           data_reduced_mapping[i].end(), data[i][j]);
        unsigned int idx = indx - data_reduced_mapping[i].begin();
        data_reduced[i][j] = idx;
      }
    }

#pragma omp parallel for
    for (size_t i = 0; i < columns_category; ++i) {
      unsigned int m =
        *std::max_element(data_categories[i].begin(), data_categories[i].end());
      category_size[i] = 32 - __builtin_clz(m);
    }

    for (size_t i = 0; i < columns_dense && verbose; ++i) {
      printf("feature %lu has been reduced to %u bits \n", i, reduced_size[i]);
    }
    max_reduced_size =
      *std::max_element(reduced_size.begin(), reduced_size.end());
    if (verbose) printf("max feature size %u \n", max_reduced_size);

    if (columns_category == 0)
      max_feature_size = max_reduced_size;
    else {
      max_category_size =
        *std::max_element(category_size.begin(), category_size.end());
      max_feature_size = std::max(max_reduced_size, max_category_size);
    }
    _init = true;
  }
}

void DataMatrix::UpdateGrad() {}
void DataMatrix::TransferToGPU(size_t free, bool verbose) {
  size_t data_size = sizeof(float) * rows;
  size_t copy_count = std::min(free / data_size, columns_dense);
  for (size_t i = 0; i < copy_count; ++i) {
    sorted_data_device[i].resize(rows);
    thrust::copy(data_reduced[i].begin(), data_reduced[i].end(),
                 sorted_data_device[i].begin());
  }
  if (verbose)
    printf("copied features data %ld from %ld \n", copy_count, columns_dense);

  free -= copy_count * data_size;

  copy_count = 0;

  for (size_t i = 0; i < columns_category; ++i) {
    if (rows * sizeof(unsigned int) < free) {
      copy_count++;
      data_category_device[i].resize(rows);
      thrust::copy(data_categories[i].begin(), data_categories[i].end(),
                   data_category_device[i].begin());
      free -= rows * sizeof(unsigned int);
    } else {
      break;
    }
  }
  if (verbose)
    printf("copied category features %ld from %ld \n", copy_count,
           columns_category);
}
}  // namespace io
}  // namespace arboretum
