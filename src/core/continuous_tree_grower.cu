#include "hip/hip_runtime.h"
#include "continuous_tree_grower.h"
#include "cuda_helpers.h"

namespace arboretum {
namespace core {
using thrust::device_vector;
using thrust::host_vector;

template <typename SUM_T, typename NODE_VALUE_T>
__global__ void gain_kernel(
  const SUM_T *const __restrict__ left_sum,
  const NODE_VALUE_T *const __restrict__ segments_fvalues, const unsigned span,
  const SUM_T *const __restrict__ parent_sum_iter,
  const unsigned int *const __restrict__ parent_count_iter, const size_t n,
  const GainFunctionParameters parameters, my_atomics *res) {
  for (unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    if (i == 0) continue;

    const unsigned int fvalue = segments_fvalues[i];
    const unsigned int fvalue_prev = segments_fvalues[i - 1];
    if (fvalue != fvalue_prev) {
      // TODO: Binary search?
      unsigned segment = 0;
      while (i >= parent_count_iter[segment + 1]) {
        segment++;
      }

      if (i == parent_count_iter[segment + 1]) continue;
      const SUM_T left_sum_offset = parent_sum_iter[segment];
      const SUM_T left_sum_value = left_sum[i] - left_sum_offset;

      const size_t left_count_offset = parent_count_iter[segment];
      const size_t left_count_value = i - left_count_offset;

      const SUM_T total_sum = parent_sum_iter[segment + 1] - left_sum_offset;
      const size_t total_count =
        parent_count_iter[segment + 1] - left_count_offset;

      const float gain = gain_func(left_sum_value, total_sum, left_count_value,
                                   total_count, parameters);
      if (gain > 0.0) {
        updateAtomicMax(&(res[segment].ulong), gain, i);
      }
    }
  }
}

template <typename NODE_T, typename SUM_T>
__global__ void filter_apply_candidates(
  float *gain, int *features, SUM_T *sum, unsigned *split, unsigned *count,
  unsigned *node_size_prefix_sum_next, SUM_T *node_sum_prefix_sum_next,
  const my_atomics *candidates, const SUM_T *split_sum, const unsigned *fvalue,
  const unsigned *fvalue_sorted, NODE_T *row2Node,
  const unsigned *node_size_prefix_sum, const SUM_T *node_sum_prefix_sum,
  const int feature, const unsigned level, const unsigned n) {
  for (unsigned i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    const unsigned node_start = node_size_prefix_sum[i];
    const unsigned node_end = node_size_prefix_sum[i + 1];
    const unsigned node_size = node_end - node_start;
    const float gain_ = candidates[i].floats[0];
    const unsigned idx = candidates[i].ints[1];
    const SUM_T node_start_sum = node_sum_prefix_sum[i];
    const SUM_T node_end_sum = node_sum_prefix_sum[i + 1];
    if (node_size > 0) {
      if (gain[i] < gain_) {
        const SUM_T split_sum_value = split_sum[idx];
        gain[i] = gain_;
        features[i] = feature;
        sum[i] = split_sum_value - node_start_sum;
        count[i] = idx - node_start;
        unsigned threshold = fvalue_sorted[idx];
        split[i] = threshold;

        unsigned block_size = MAX_THREADS > node_size ? node_size : MAX_THREADS;
        unsigned grid_size =
          unsigned((node_size + block_size - 1) / block_size);
        hipStream_t s;
        DEVICE_OK(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
        apply_split<NODE_T><<<grid_size, block_size, 0, s>>>(
          row2Node + node_start, fvalue + node_start, threshold, level,
          node_size);
        DEVICE_OK(hipDeviceSynchronize());
        DEVICE_OK(hipStreamDestroy(s));
        node_size_prefix_sum_next[2 * i + 1] = idx;
        node_size_prefix_sum_next[2 * i + 2] = node_end;
        node_sum_prefix_sum_next[2 * i + 1] = split_sum_value;
        node_sum_prefix_sum_next[2 * i + 2] = node_end_sum;
      } else if (gain[i] == 0 && features[i] == -1) {
        sum[i] = node_end_sum - node_start_sum;
        split[i] = (unsigned)-1;
        count[i] = node_size;
        node_size_prefix_sum_next[2 * i + 1] =
          node_size_prefix_sum_next[2 * i + 2] = node_end;
        node_sum_prefix_sum_next[2 * i + 1] =
          node_sum_prefix_sum_next[2 * i + 2] = node_end_sum;
      }
    } else {
      node_size_prefix_sum_next[2 * i + 1] =
        node_size_prefix_sum_next[2 * i + 2] = node_end;
      node_sum_prefix_sum_next[2 * i + 1] =
        node_sum_prefix_sum_next[2 * i + 2] = node_end_sum;
    }
  }
}

template <typename NODE_T, typename GRAD_T, typename SUM_T>
ContinuousTreeGrower<NODE_T, GRAD_T, SUM_T>::ContinuousTreeGrower(
  const size_t size, const unsigned depth, const unsigned hist_size,
  const BestSplit<SUM_T> *best, Histogram<SUM_T> *features_histogram,
  const InternalConfiguration *config)
    : BaseGrower<NODE_T, GRAD_T, SUM_T>(size, depth, best, features_histogram,
                                        config) {
  node_fvalue.resize(size);
  node_fvalue_sorted.resize(size);
  sum.resize(size);
  run_lenght.resize(1);

  size_t temp_storage_bytes = 0;

  OK(hipcub::DeviceSegmentedRadixSort::SortPairs(
    NULL, temp_storage_bytes, (NODE_T *)nullptr, (NODE_T *)nullptr,
    (GRAD_T *)nullptr, (GRAD_T *)nullptr, size, 1 << this->depth,
    (unsigned *)nullptr, (unsigned *)nullptr, 0, 1));

  this->temp_bytes_allocated =
    std::max(this->temp_bytes_allocated, temp_storage_bytes);

  temp_storage_bytes = 0;

  SUM_T initial_value;
  init(initial_value);
  hipcub::Sum sum_op;

  OK(hipcub::DeviceScan::ExclusiveScan(NULL, temp_storage_bytes, (GRAD_T *)nullptr,
                                    (SUM_T *)nullptr, sum_op, initial_value,
                                    size));

  this->temp_bytes_allocated =
    std::max(this->temp_bytes_allocated, temp_storage_bytes);

  temp_storage_bytes = 0;

  OK(hipcub::DeviceReduce::ReduceByKey(
    NULL, temp_storage_bytes, (NODE_T *)nullptr, (NODE_T *)nullptr,
    (GRAD_T *)nullptr, (SUM_T *)nullptr,
    thrust::raw_pointer_cast(run_lenght.data()), sum_op, size));

  this->temp_bytes_allocated =
    std::max(this->temp_bytes_allocated, temp_storage_bytes);

  temp_storage_bytes = 0;

  OK(hipcub::DeviceRunLengthEncode::Encode(
    NULL, temp_storage_bytes, (NODE_T *)nullptr, (NODE_T *)nullptr,
    (NODE_T *)nullptr, thrust::raw_pointer_cast(run_lenght.data()), size));

  this->temp_bytes_allocated =
    std::max(this->temp_bytes_allocated, temp_storage_bytes);

  OK(hipMalloc(&this->temp_bytes, this->temp_bytes_allocated));
}

template <typename NODE_T, typename GRAD_T, typename SUM_T>
inline void ContinuousTreeGrower<NODE_T, GRAD_T, SUM_T>::ApplySplit(
  NODE_T *row2Node, const unsigned level, const unsigned threshold, size_t from,
  size_t to) {
  int gridSize;
  int blockSize;
  compute1DInvokeConfig(to - from, &gridSize, &blockSize, apply_split<NODE_T>);

  apply_split<NODE_T><<<gridSize, blockSize, 0, this->stream>>>(
    row2Node + from,
    ((unsigned *)thrust::raw_pointer_cast(node_fvalue.data())) + from,
    threshold, level, to - from);
}

template <typename NODE_T, typename GRAD_T, typename SUM_T>
template <typename NODE_VALUE_T>
inline void ContinuousTreeGrower<NODE_T, GRAD_T, SUM_T>::ProcessDenseFeature(
  const device_vector<NODE_T> &row2Node, const device_vector<GRAD_T> &grad_d,
  unsigned int *fvalue_d, unsigned int *fvalue_h,
  const device_vector<SUM_T> &parent_node_sum,
  const device_vector<unsigned int> &parent_node_count,
  const unsigned char fvalue_size, const unsigned level, const unsigned depth,
  const GainFunctionParameters gain_param, const bool partition_only,
  const int fid) {
  const unsigned lenght = 1 << level;

  OK(hipMemsetAsync(thrust::raw_pointer_cast(this->result_d.data()), 0,
                     lenght * sizeof(my_atomics), this->stream));

  unsigned int *fvalue_tmp = NULL;

  if (fvalue_d != nullptr) {
    fvalue_tmp = fvalue_d;
  } else {
    OK(hipMemcpyAsync(thrust::raw_pointer_cast(this->fvalue.data()), fvalue_h,
                       this->size * sizeof(unsigned int),
                       hipMemcpyHostToDevice, this->stream));
    fvalue_tmp = thrust::raw_pointer_cast(this->fvalue.data());
  }

  if (level != 0) {
    const unsigned lenght = 1 << (level - 1);
    int gridSize = 0;
    int blockSize = 0;

    compute1DInvokeConfig(lenght, &gridSize, &blockSize,
                          partition<NODE_T, unsigned>, 0, 1);
    partition<NODE_T, unsigned, 2><<<gridSize, blockSize, 0, this->stream>>>(
      (unsigned *)thrust::raw_pointer_cast(node_fvalue.data()),
      thrust::raw_pointer_cast(row2Node.data()), fvalue_tmp,
      thrust::raw_pointer_cast(parent_node_count.data()), depth - level - 1,
      this->temp_bytes_allocated, this->temp_bytes, this->size, lenght);

    OK(hipEventRecord(this->event, this->stream));

    OK(hipStreamWaitEvent(this->copy_d2h_stream, this->event, 0));

    OK(hipMemcpyAsync(fvalue_h, thrust::raw_pointer_cast(node_fvalue.data()),
                       this->size * sizeof(unsigned int),
                       hipMemcpyDeviceToHost, this->copy_d2h_stream));

    if (fvalue_d != nullptr) {
      OK(hipMemcpyAsync(fvalue_d, thrust::raw_pointer_cast(node_fvalue.data()),
                         this->size * sizeof(unsigned int),
                         hipMemcpyDeviceToDevice, this->copy_d2h_stream));
    }

    this->d_fvalue_partitioned =
      (unsigned *)thrust::raw_pointer_cast(node_fvalue.data());

  } else {
    this->d_fvalue_partitioned = fvalue_tmp;
  }

  if (partition_only) return;

  // FIXME: fvalue_size + 1 or just fvalue_size?
  CubDebugExit(hipcub::DeviceSegmentedRadixSort::SortPairs(
    this->temp_bytes, this->temp_bytes_allocated, this->d_fvalue_partitioned,
    (unsigned *)thrust::raw_pointer_cast(node_fvalue_sorted.data()),
    thrust::raw_pointer_cast(grad_d.data()),
    thrust::raw_pointer_cast(this->grad_sorted.data()), this->size, 1 << level,
    thrust::raw_pointer_cast(parent_node_count.data()),
    thrust::raw_pointer_cast(parent_node_count.data()) + 1, 0, fvalue_size + 1,
    this->stream));

  SUM_T initial_value;
  init(initial_value);
  hipcub::Sum sum_op;

  OK(hipcub::DeviceScan::ExclusiveScan(
    this->temp_bytes, this->temp_bytes_allocated,
    thrust::raw_pointer_cast(this->grad_sorted.data()),
    thrust::raw_pointer_cast(sum.data()), sum_op, initial_value, this->size,
    this->stream));

  gain_kernel<<<this->gridSizeGain, this->blockSizeGain, 0, this->stream>>>(
    thrust::raw_pointer_cast(sum.data()),
    (unsigned *)thrust::raw_pointer_cast(node_fvalue_sorted.data()), lenght,
    thrust::raw_pointer_cast(parent_node_sum.data()),
    thrust::raw_pointer_cast(parent_node_count.data()), this->size, gain_param,
    thrust::raw_pointer_cast(this->result_d.data()));
}

template <typename NODE_T, typename GRAD_T, typename SUM_T>
inline void ContinuousTreeGrower<NODE_T, GRAD_T, SUM_T>::FindBest(
  BestSplit<SUM_T> &best, device_vector<NODE_T> &row2Node,
  const device_vector<SUM_T> &parent_node_sum,
  const device_vector<unsigned int> &parent_node_count, const unsigned fid,
  const unsigned level, const unsigned depth, const unsigned size) {
  int gridSize = 0;
  int blockSize = 0;

  compute1DInvokeConfig(size, &gridSize, &blockSize,
                        filter_apply_candidates<NODE_T, SUM_T>);

  filter_apply_candidates<NODE_T, SUM_T>
    <<<gridSize, blockSize, 0, this->stream>>>(
      thrust::raw_pointer_cast(best.gain.data()),
      thrust::raw_pointer_cast(best.feature.data()),
      thrust::raw_pointer_cast(best.sum.data()),
      thrust::raw_pointer_cast(best.split_value.data()),
      thrust::raw_pointer_cast(best.count.data()),
      thrust::raw_pointer_cast(best.parent_node_count_next.data()),
      thrust::raw_pointer_cast(best.parent_node_sum_next.data()),
      thrust::raw_pointer_cast(this->result_d.data()),
      thrust::raw_pointer_cast(this->sum.data()), this->d_fvalue_partitioned,
      (unsigned *)thrust::raw_pointer_cast(this->node_fvalue_sorted.data()),
      thrust::raw_pointer_cast(row2Node.data()),
      thrust::raw_pointer_cast(parent_node_count.data()),
      thrust::raw_pointer_cast(parent_node_sum.data()), fid, depth - level - 2,
      size);
}

template class ContinuousTreeGrower<unsigned, float, float>;

template void ContinuousTreeGrower<unsigned, float, float>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned> &row2Node,
            const device_vector<float> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h, const device_vector<float> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void ContinuousTreeGrower<unsigned, float, float>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned> &row2Node,
                 const device_vector<float> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<float> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned, float, double>;

template void
ContinuousTreeGrower<unsigned, float, double>::ProcessDenseFeature<unsigned>(
  const device_vector<unsigned> &row2Node, const device_vector<float> &grad_d,
  unsigned int *fvalue_d, unsigned int *fvalue_h,
  const device_vector<double> &parent_node_sum,
  const device_vector<unsigned int> &parent_node_count,
  const unsigned char fvalue_size, const unsigned level, const unsigned depth,
  const GainFunctionParameters gain_param, const bool partition_only,
  const int fid);

template void
ContinuousTreeGrower<unsigned, float, double>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned> &row2Node,
                 const device_vector<float> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<double> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned, float2, float2>;

template void
ContinuousTreeGrower<unsigned, float2, float2>::ProcessDenseFeature<unsigned>(
  const device_vector<unsigned> &row2Node, const device_vector<float2> &grad_d,
  unsigned int *fvalue_d, unsigned int *fvalue_h,
  const device_vector<float2> &parent_node_sum,
  const device_vector<unsigned int> &parent_node_count,
  const unsigned char fvalue_size, const unsigned level, const unsigned depth,
  const GainFunctionParameters gain_param, const bool partition_only,
  const int fid);

template void
ContinuousTreeGrower<unsigned, float2, float2>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned> &row2Node,
                 const device_vector<float2> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<float2> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned, float2, mydouble2>;

template void
ContinuousTreeGrower<unsigned, float2, mydouble2>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned> &row2Node,
            const device_vector<float2> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h,
            const device_vector<mydouble2> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void
ContinuousTreeGrower<unsigned, float2, mydouble2>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned> &row2Node,
                 const device_vector<float2> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<mydouble2> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned short, float, float>;
template class ContinuousTreeGrower<unsigned short, float, double>;
template class ContinuousTreeGrower<unsigned short, float2, float2>;
template class ContinuousTreeGrower<unsigned short, float2, mydouble2>;

template class ContinuousTreeGrower<unsigned char, float, float>;
template class ContinuousTreeGrower<unsigned char, float, double>;
template class ContinuousTreeGrower<unsigned char, float2, float2>;
template class ContinuousTreeGrower<unsigned char, float2, mydouble2>;

template class ContinuousTreeGrower<unsigned long, float, float>;

template void
ContinuousTreeGrower<unsigned long, float, float>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned long> &row2Node,
            const device_vector<float> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h, const device_vector<float> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void
ContinuousTreeGrower<unsigned long, float, float>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned long> &row2Node,
                 const device_vector<float> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<float> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned long, float, double>;

template void
ContinuousTreeGrower<unsigned long, float, double>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned long> &row2Node,
            const device_vector<float> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h,
            const device_vector<double> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void
ContinuousTreeGrower<unsigned long, float, double>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned long> &row2Node,
                 const device_vector<float> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<double> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned long, float2, float2>;

template void
ContinuousTreeGrower<unsigned long, float2, float2>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned long> &row2Node,
            const device_vector<float2> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h,
            const device_vector<float2> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void
ContinuousTreeGrower<unsigned long, float2, float2>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned long> &row2Node,
                 const device_vector<float2> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<float2> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

template class ContinuousTreeGrower<unsigned long, float2, mydouble2>;

template void
ContinuousTreeGrower<unsigned long, float2, mydouble2>::ProcessDenseFeature<
  unsigned>(const device_vector<unsigned long> &row2Node,
            const device_vector<float2> &grad_d, unsigned int *fvalue_d,
            unsigned int *fvalue_h,
            const device_vector<mydouble2> &parent_node_sum,
            const device_vector<unsigned int> &parent_node_count,
            const unsigned char fvalue_size, const unsigned level,
            const unsigned depth, const GainFunctionParameters gain_param,
            const bool partition_only, const int fid);

template void
ContinuousTreeGrower<unsigned long, float2, mydouble2>::ProcessDenseFeature<
  unsigned long>(const device_vector<unsigned long> &row2Node,
                 const device_vector<float2> &grad_d, unsigned int *fvalue_d,
                 unsigned int *fvalue_h,
                 const device_vector<mydouble2> &parent_node_sum,
                 const device_vector<unsigned int> &parent_node_count,
                 const unsigned char fvalue_size, const unsigned level,
                 const unsigned depth, const GainFunctionParameters gain_param,
                 const bool partition_only, const int fid);

}  // namespace core
}  // namespace arboretum
