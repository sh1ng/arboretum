#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <algorithm>
#include <ctime>
#include <limits>
#include <random>
#include "best_splits.h"
#include "builder.h"
#include "hipcub/hipcub.hpp"
#include "cuda_helpers.h"
#include "garden.h"
#include "histogram.h"
#include "objective.h"
#include "param.h"

namespace arboretum {
namespace core {
using namespace thrust;
using namespace thrust::cuda;
using thrust::device_vector;
using thrust::host_vector;

template <typename SUM_T, typename NODE_T>
__global__ void update_by_last_tree(float *y, const SUM_T *best_sum,
                                    const unsigned *best_count,
                                    const SUM_T *sum_prefix_sum,
                                    const unsigned *count_prefix_sum,
                                    const NODE_T *row2Node,
                                    const TreeParam param, const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    NODE_T leaf = row2Node[i];

    unsigned segment = 0;
    while (i >= count_prefix_sum[segment + 1]) {
      segment++;
    }

    float delta = 0.0;
    const SUM_T left_sum = best_sum[segment];
    const SUM_T right_sum =
      sum_prefix_sum[segment + 1] - sum_prefix_sum[segment] - left_sum;

    const unsigned left_count = best_count[segment];
    const unsigned right_count =
      count_prefix_sum[segment + 1] - count_prefix_sum[segment] - left_count;

    if (leaf % 2 == 0) {
      delta = Weight(left_sum, left_count, param) * param.eta;
    } else {
      delta = Weight(right_sum, right_count, param) * param.eta;
    }
    assert(isfinite(delta));

    y[i] += delta;
  }
}

template <typename NODE_T, typename GRAD_T, typename SUM_T,
          typename TREE_GROWER>
class ContinuousGardenBuilder : public GardenBuilderBase {
 public:
  ContinuousGardenBuilder(const TreeParam &param, io::DataMatrix *data,
                          const InternalConfiguration &config,
                          ApproximatedObjective<GRAD_T> *objective,
                          const bool verbose)
      : verbose(verbose),
        rnd(config.seed),
        overlap_depth(config.overlap),
        param(param),
        gain_param(param.min_leaf_size, param.min_child_weight,
                   param.gamma_absolute, param.gamma_relative, param.lambda,
                   param.alpha),
        objective(objective),
        best(1 << param.depth, param.hist_size),
        features_histograms(1 << param.depth, param.hist_size,
                            data->columns_dense) {
    active_fids.resize(data->columns);

    row2Node.resize(data->rows, 0);
    _bestSplit.resize(1 << (param.depth - 2));
    _nodeStat.resize(1 << (param.depth - 2));

    growers = new TREE_GROWER *[overlap_depth];

    for (size_t i = 0; i < overlap_depth; ++i) {
      growers[i] = new TREE_GROWER(data->rows, param.depth, param.hist_size,
                                   &best, &features_histograms, &config);
    }
  }

  virtual ~ContinuousGardenBuilder() {
    for (auto i = 0; i < overlap_depth; ++i) {
      delete growers[i];
    }

    delete[] growers;
  }

  virtual void InitGrowingTree(const size_t columns) override {
    int take = (int)(param.colsample_bytree * columns);
    if (take == 0) {
      printf("colsample_bytree is too small %f for %ld columns \n",
             param.colsample_bytree, columns);
      throw "colsample_bytree is too small";
    }
    take = (int)(param.colsample_bytree * param.colsample_bylevel * columns);
    if (take == 0) {
      printf(
        "colsample_bytree and colsample_bylevel are too small %f %f for "
        "%ld columns \n",
        param.colsample_bytree, param.colsample_bylevel, columns);
      throw "colsample_bytree and colsample_bylevel are too small";
    }

    for (size_t i = 0; i < columns; ++i) {
      active_fids[i] = i;
    }

    shuffle(active_fids.begin(), active_fids.end(), rnd);

    thrust::fill(row2Node.begin(), row2Node.end(), 0);
    for (size_t i = 0; i < _nodeStat.size(); ++i) {
      _nodeStat[i].Clean();
    }
    for (size_t i = 0; i < _bestSplit.size(); ++i) {
      _bestSplit[i].Clean();
    }
    this->features_histograms.Clear();
    OK(hipDeviceSynchronize());
    for (size_t i = 0; i < overlap_depth; i++) {
      OK(hipStreamSynchronize(growers[i]->stream));
    }
  }

  virtual void InitTreeLevel(const int level, const size_t columns) override {
    int take = (int)(param.colsample_bytree * columns);
    shuffle(active_fids.begin(), active_fids.begin() + take, rnd);
  }

  virtual void GrowTree(RegTree *tree, io::DataMatrix *data,
                        const unsigned short label) override {
    grad_slice = const_cast<GRAD_T *>(
      thrust::raw_pointer_cast(objective->grad.data() + label * data->rows));

    InitGrowingTree(data->columns);

    for (unsigned int i = 0; (i + 1) < param.depth; ++i) {
      InitTreeLevel(i, data->columns);
      UpdateNodeStat(i, data, tree);
      FindBestSplits(i, data);
      UpdateTree(i, tree, data);
    }

    for (size_t i = 0; i < overlap_depth; i++) {
      OK(hipStreamSynchronize(growers[i]->stream));
    }

    OK(hipDeviceSynchronize());
    OK(hipGetLastError());

    UpdateLeafWeight(tree);
    for (size_t i = 0; i < overlap_depth; i++) {
      OK(hipStreamSynchronize(growers[i]->stream));
    }

    OK(hipDeviceSynchronize());
    OK(hipGetLastError());

    UpdateByLastTree(data);
  }

  void UpdateByLastTree(io::DataMatrix *data) {
    int gridSize = 0;
    int blockSize = 0;
    compute1DInvokeConfig(data->rows, &gridSize, &blockSize,
                          update_by_last_tree<SUM_T, NODE_T>);

    update_by_last_tree<SUM_T, NODE_T><<<gridSize, blockSize>>>(
      thrust::raw_pointer_cast(data->y_internal_d.data()),
      thrust::raw_pointer_cast(this->best.sum.data()),
      thrust::raw_pointer_cast(this->best.count.data()),
      thrust::raw_pointer_cast(this->best.parent_node_sum.data()),
      thrust::raw_pointer_cast(this->best.parent_node_count.data()),
      thrust::raw_pointer_cast(row2Node.data()), param, data->rows);
  }

  virtual void PredictByGrownTree(
    RegTree *tree, io::DataMatrix *data,
    thrust::host_vector<float> &out) const override {
    // tree->Predict(data, _rowIndex2Node, out);
  }

 private:
  bool verbose;
  std::default_random_engine rnd;
  std::vector<unsigned int> active_fids;
  const unsigned short overlap_depth;
  const TreeParam param;
  const GainFunctionParameters gain_param;
  GRAD_T *grad_slice;
  ApproximatedObjective<GRAD_T> *objective;
  std::vector<NodeStat<SUM_T>> _nodeStat;
  std::vector<Split<SUM_T>> _bestSplit;

  device_vector<NODE_T> row2Node;
  size_t temp_bytes_per_rec = 0;

  TREE_GROWER **growers;
  BestSplit<SUM_T> best;
  Histogram<SUM_T> features_histograms;

  void FindBestSplits(const unsigned int level, io::DataMatrix *data) {
    unsigned length = 1 << level;

    unsigned int take = (unsigned int)(param.colsample_bylevel *
                                       param.colsample_bytree * data->columns);

    growers[0]->template Partition<GRAD_T, 1>(
      thrust::raw_pointer_cast(objective->grad.data()),
      thrust::raw_pointer_cast(row2Node.data()), this->best.parent_node_count,
      level, param.depth);
    growers[0]->template Partition<float, 1>(
      thrust::raw_pointer_cast(data->y_internal_d.data()),
      thrust::raw_pointer_cast(row2Node.data()), this->best.parent_node_count,
      level, param.depth);

    growers[0]->template Partition<float, 1>(
      thrust::raw_pointer_cast(data->y_hat_d.data()),
      thrust::raw_pointer_cast(row2Node.data()), this->best.parent_node_count,
      level, param.depth);

    OK(hipStreamSynchronize(growers[0]->stream));
    OK(hipStreamSynchronize(growers[0]->copy_d2h_stream));

    if (level != 0) {
      this->best.NextLevel(length);
    }

    for (size_t j = 0; j < data->columns; ++j) {
      for (size_t i = 0; i < overlap_depth && (j + i) < data->columns; ++i) {
        if (j != 0 && (i + 1) < overlap_depth) {
          continue;
        }

        size_t active_fid = active_fids[j + i];
        size_t circular_fid = (j + i) % overlap_depth;

        if (active_fid < data->columns_dense) {
          ProcessDenseFeature(active_fid, circular_fid, level, data,
                              (j + i) >= take);
        } else {
          ProcessCategoryFeature(active_fid - data->columns_dense, circular_fid,
                                 level, data);
        }
      }

      size_t circular_fid = j % overlap_depth;

      if (active_fids[j] < data->columns_dense) {
        // FIXME:
        // if ((data->reduced_size[active_fids[j]] + level) <
        //     sizeof(unsigned char) * CHAR_BIT) {
        //   GetBestSplitForDenseFeature<unsigned char>(
        //       active_fids[j], circular_fid, level,
        //       data->data_reduced_mapping[active_fids[j]],
        //       data->reduced_size[active_fids[j]]);
        // } else if ((data->reduced_size[active_fids[j]] + level) <
        //            sizeof(unsigned short) * CHAR_BIT) {
        //   GetBestSplitForDenseFeature<unsigned short>(
        //       active_fids[j], circular_fid, level,
        //       data->data_reduced_mapping[active_fids[j]],
        //       data->reduced_size[active_fids[j]]);
        // } else if ((data->reduced_size[active_fids[j]] + level) <
        //            sizeof(unsigned int) * CHAR_BIT) {
        GetBestSplitForDenseFeature<unsigned int>(
          active_fids[j], circular_fid, level,
          data->data_reduced_mapping[active_fids[j]],
          data->reduced_size[active_fids[j]], j >= take);
        // } else {
        //   GetBestSplitForDenseFeature<NODE_T>(
        //       active_fids[j], circular_fid, level,
        //       data->data_reduced_mapping[active_fids[j]],
        //       data->reduced_size[active_fids[j]]);
        // }
      } else {
        if ((data->category_size[active_fids[j] - data->columns_dense] +
             level) < sizeof(unsigned char) * CHAR_BIT) {
          GetBestSplitForCategoryFeature<unsigned char>(
            active_fids[j] - data->columns_dense, data->columns_dense,
            circular_fid, length, data);
        } else if ((data->category_size[active_fids[j] - data->columns_dense] +
                    level) < sizeof(unsigned short) * CHAR_BIT) {
          GetBestSplitForCategoryFeature<unsigned short>(
            active_fids[j] - data->columns_dense, data->columns_dense,
            circular_fid, length, data);
        } else if ((data->category_size[active_fids[j] - data->columns_dense] +
                    level) < sizeof(unsigned int) * CHAR_BIT) {
          GetBestSplitForCategoryFeature<unsigned int>(
            active_fids[j] - data->columns_dense, data->columns_dense,
            circular_fid, length, data);
        } else {
          GetBestSplitForCategoryFeature<NODE_T>(
            active_fids[j] - data->columns_dense, data->columns_dense,
            circular_fid, length, data);
        }
      }
    }

    // growers[0]->template Partition<GRAD_T, 1>(
    //   thrust::raw_pointer_cast(grad_d.data()),
    //   thrust::raw_pointer_cast(row2Node.data()), parent_node_count, level,
    //   param.depth);
  }

  // FIXME: use template
  template <typename NODE_VALUE_T>
  inline void GetBestSplitForDenseFeature(
    const int active_fid, const size_t circular_fid, const unsigned level,
    const std::vector<float> &data_reduced_mapping, const unsigned reduced_size,
    const bool partition_only) {
    if (!partition_only) {
      const unsigned length = 1 << level;

      //   OK(hipStreamSynchronize(growers[circular_fid]->stream));

      growers[circular_fid]->FindBest(
        this->best, this->row2Node, this->best.parent_node_sum,
        this->best.parent_node_count, active_fid, level, param.depth, length);
    }
    OK(hipStreamSynchronize(growers[circular_fid]->stream));
    OK(hipStreamSynchronize(growers[circular_fid]->copy_d2h_stream));
  }

  template <typename NODE_VALUE_T>
  inline void GetBestSplitForCategoryFeature(const int active_fid,
                                             const size_t columns_dense,
                                             const size_t circular_fid,
                                             const size_t lenght,
                                             const io::DataMatrix *data) {}

  void ProcessDenseFeature(const size_t active_fid, const size_t circular_fid,
                           const size_t level, io::DataMatrix *data,
                           const bool partition_only) {
    // if ((data->reduced_size[active_fid]) < sizeof(unsigned char) *
    // CHAR_BIT)
    // {
    //   growers[circular_fid]->template ProcessDenseFeature<unsigned char>(
    //       row2Node, grad_d,
    //       data->sorted_data_device[active_fid].size() > 0
    //           ? thrust::raw_pointer_cast(
    //                 data->sorted_data_device[active_fid].data())
    //           : nullptr,
    //       thrust::raw_pointer_cast(data->data_reduced[active_fid].data()),
    //       parent_node_sum, parent_node_count,
    //       data->reduced_size[active_fid], level, gain_param);
    // } else if ((data->reduced_size[active_fid]) <
    //            sizeof(unsigned short) * CHAR_BIT) {
    //   growers[circular_fid]->template ProcessDenseFeature<unsigned short>(
    //       row2Node, grad_d,
    //       data->sorted_data_device[active_fid].size() > 0
    //           ? thrust::raw_pointer_cast(
    //                 data->sorted_data_device[active_fid].data())
    //           : nullptr,
    //       thrust::raw_pointer_cast(data->data_reduced[active_fid].data()),
    //       parent_node_sum, parent_node_count,
    //       data->reduced_size[active_fid], level, gain_param);
    // } else if ((data->reduced_size[active_fid]) <
    //            sizeof(unsigned int) * CHAR_BIT) {
    //   growers[circular_fid]->template ProcessDenseFeature<unsigned int>(
    //       row2Node, grad_d,
    //       data->sorted_data_device[active_fid].size() > 0
    //           ? thrust::raw_pointer_cast(
    //                 data->sorted_data_device[active_fid].data())
    //           : nullptr,
    //       thrust::raw_pointer_cast(data->data_reduced[active_fid].data()),
    //       parent_node_sum, parent_node_count,
    //       data->reduced_size[active_fid], level, gain_param);
    // } else {
    growers[circular_fid]->template ProcessDenseFeature<NODE_T>(
      row2Node, objective->grad,
      data->sorted_data_device[active_fid].size() > 0
        ? thrust::raw_pointer_cast(data->sorted_data_device[active_fid].data())
        : nullptr,
      thrust::raw_pointer_cast(data->data_reduced[active_fid].data()),
      this->best.parent_node_sum, this->best.parent_node_count,
      data->reduced_size[active_fid], level, param.depth, gain_param,
      partition_only, active_fid);
    // }
  }

  inline void ProcessCategoryFeature(const size_t active_fid,
                                     const size_t circular_fid,
                                     const size_t level,
                                     const io::DataMatrix *data) {
    if ((data->category_size[active_fid] + level) <
        sizeof(unsigned char) * CHAR_BIT) {
      growers[circular_fid]->template ProcessCategoryFeature<unsigned char>(
        row2Node, objective->grad, data->data_category_device[active_fid],
        data->data_categories[active_fid], this->best.parent_node_sum,
        this->best.parent_node_count, data->category_size[active_fid], level,
        gain_param);
    } else if ((data->category_size[active_fid] + level) <
               sizeof(unsigned short) * CHAR_BIT) {
      growers[circular_fid]->template ProcessCategoryFeature<unsigned short>(
        row2Node, objective->grad, data->data_category_device[active_fid],
        data->data_categories[active_fid], this->best.parent_node_sum,
        this->best.parent_node_count, data->category_size[active_fid], level,
        gain_param);
    } else if ((data->category_size[active_fid] + level) <
               sizeof(unsigned int) * CHAR_BIT) {
      growers[circular_fid]->template ProcessCategoryFeature<unsigned int>(
        row2Node, objective->grad, data->data_category_device[active_fid],
        data->data_categories[active_fid], this->best.parent_node_sum,
        this->best.parent_node_count, data->category_size[active_fid], level,
        gain_param);
    } else {
      growers[circular_fid]->template ProcessCategoryFeature<NODE_T>(
        row2Node, objective->grad, data->data_category_device[active_fid],
        data->data_categories[active_fid], this->best.parent_node_sum,
        this->best.parent_node_count, data->category_size[active_fid], level,
        gain_param);
    }
  }

  void UpdateNodeStat(const int level, const io::DataMatrix *data,
                      const RegTree *tree) {
    const unsigned len = 1 << level;
    best.Clear(len);

    if (level == 0) {
      SUM_T zero;
      init(zero);

      best.parent_node_count[0] = 0;
      best.parent_node_count[1] = unsigned(data->rows);
      best.parent_node_sum[0] = zero;

      OK(hipcub::DeviceReduce::Sum(
        this->growers[0]->temp_bytes, this->growers[0]->temp_bytes_allocated,
        thrust::raw_pointer_cast(objective->grad.data()),
        thrust::raw_pointer_cast(&best.parent_node_sum[1]), data->rows));

      CubDebugExit(hipDeviceSynchronize());
    }

    for (unsigned i = 0; i < len; ++i) {
      _nodeStat[i].gain =
        0.0;  // todo: gain_func(_nodeStat[i].count, _nodeStat[i].sum_grad);
      _bestSplit[i].Clean();
    }
  }

  void UpdateTree(const int level, RegTree *tree, io::DataMatrix *data) {
    const unsigned int offset = Node::HeapOffset(level);
    const size_t len = 1 << level;
    CubDebugExit(hipDeviceSynchronize());

    best.Sync(1 << level);
    CubDebugExit(hipDeviceSynchronize());

    for (unsigned i = 0; i < len; ++i) {
      const unsigned quantized = best.split_value_h[i];

      _bestSplit[i].quantized = quantized;
      _bestSplit[i].count = best.count_h[i];
      _bestSplit[i].fid = best.feature_h[i];
      _bestSplit[i].sum_grad = best.sum_h[i];
      if (best.feature_h[i] != -1) {
        _bestSplit[i].split_value =
          quantized >= data->data_reduced_mapping[best.feature_h[i]].size()
            ? std::numeric_limits<float>::infinity()
            : data->data_reduced_mapping[best.feature_h[i]][quantized];
      } else {
        _bestSplit[i].gain = 0.0;
        _bestSplit[i].fid = 0;
        _bestSplit[i].split_value = std::numeric_limits<float>::infinity();
        _bestSplit[i].count =
          best.parent_node_count_h[i + 1] - best.parent_node_count_h[i];
        _bestSplit[i].sum_grad =
          best.parent_node_sum_h[i + 1] - best.parent_node_sum_h[i];
      }
      const Split<SUM_T> &best = _bestSplit[i];
      tree->nodes[i + offset].threshold = best.split_value;
      tree->nodes[i + offset].category = best.category;
      tree->nodes[i + offset].fid = best.fid < 0 ? 0 : best.fid;
      tree->nodes[i + offset].quantized = best.quantized;
    }
  }

  void UpdateLeafWeight(RegTree *tree) const {
    const unsigned int offset_1 = Node::HeapOffset(tree->depth - 2);
    const unsigned int offset = Node::HeapOffset(tree->depth - 1);

    for (unsigned int i = 0, len = (1 << (tree->depth - 2)); i < len; ++i) {
      const Split<SUM_T> &split = _bestSplit[i];
      tree->leaf_level[tree->ChildNode(i + offset_1, true) - offset] =
        split.LeafWeight(param) * param.eta;
      tree->leaf_level[tree->ChildNode(i + offset_1, false) - offset] =
        split.LeafWeight(
          this->best.parent_node_count_h[i + 1] -
            this->best.parent_node_count_h[i],
          this->best.parent_node_sum_h[i + 1] - this->best.parent_node_sum_h[i],
          param) *
        param.eta;
    }
  }
};

Garden::Garden(const TreeParam &param, const Verbose &verbose,
               const InternalConfiguration &cfg)
    : param(param), verbose(verbose), cfg(cfg), _init(false) {}

void Garden::GrowTree(io::DataMatrix *data, float *grad) {
  if (param.method == Method::Exact)
    data->InitExact(verbose.data);
  else
    data->InitHist(param.hist_size, verbose.data);

  if (!_init) {
    switch (param.objective) {
      case LinearRegression: {
        auto obj = new RegressionObjective(data, param.initial_y);

        if (data->max_feature_size + 1 <= sizeof(unsigned char) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float, double,
                ContinuousTreeGrower<unsigned, float, double>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float, double,
                HistTreeGrower<unsigned, float, double>>(param, data, cfg, obj,
                                                         verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float, float,
                ContinuousTreeGrower<unsigned, float, float>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float, float, HistTreeGrower<unsigned, float, float>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned short) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float, double,
                ContinuousTreeGrower<unsigned, float, double>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float, double,
                HistTreeGrower<unsigned, float, double>>(param, data, cfg, obj,
                                                         verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float, float,
                ContinuousTreeGrower<unsigned, float, float>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float, float, HistTreeGrower<unsigned, float, float>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned int) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned int, float, double,
                ContinuousTreeGrower<unsigned, float, double>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned int, float, double,
                HistTreeGrower<unsigned, float, double>>(param, data, cfg, obj,
                                                         verbose.booster);

          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned int, float, float,
                ContinuousTreeGrower<unsigned, float, float>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned int, float, float,
                HistTreeGrower<unsigned, float, float>>(param, data, cfg, obj,
                                                        verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned long) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long, float, double,
                ContinuousTreeGrower<unsigned long, float, double>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long, float, double,
                HistTreeGrower<unsigned long, float, double>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long, float, float,
                ContinuousTreeGrower<unsigned long, float, float>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long, float, float,
                HistTreeGrower<unsigned long, float, float>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned long long) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float, double,
                ContinuousTreeGrower<unsigned long long, float, double>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float, double,
                HistTreeGrower<unsigned long long, float, double>>(
                param, data, cfg, obj, verbose.booster);

          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float, float,
                ContinuousTreeGrower<unsigned long long, float, float>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float, float,
                HistTreeGrower<unsigned long long, float, float>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else {
          throw "unsupported dimensionality";
        }
        _objective = obj;
      }

      break;
      case LogisticRegression: {
        auto obj = new LogisticRegressionObjective(data, param.initial_y);

        if (data->max_feature_size + 1 <= sizeof(unsigned char) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, mydouble2,
                ContinuousTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, mydouble2,
                HistTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, float2,
                ContinuousTreeGrower<unsigned, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, float2,
                HistTreeGrower<unsigned, float2, float2>>(param, data, cfg, obj,
                                                          verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned short) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, mydouble2,
                ContinuousTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, mydouble2,
                HistTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, float2,
                ContinuousTreeGrower<unsigned, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned, float2, float2,
                HistTreeGrower<unsigned, float2, float2>>(param, data, cfg, obj,
                                                          verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned int) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned int, float2, mydouble2,
                ContinuousTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned int, float2, mydouble2,
                HistTreeGrower<unsigned, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned int, float2, float2,
                ContinuousTreeGrower<unsigned, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned int, float2, float2,
                HistTreeGrower<unsigned, float2, float2>>(param, data, cfg, obj,
                                                          verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned long) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, mydouble2,
                ContinuousTreeGrower<unsigned long, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, mydouble2,
                HistTreeGrower<unsigned long, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, float2,
                ContinuousTreeGrower<unsigned long, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, float2,
                HistTreeGrower<unsigned long, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else if (data->max_feature_size + 1 <=
                   sizeof(unsigned long long) * CHAR_BIT) {
          if (cfg.double_precision) {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, mydouble2,
                ContinuousTreeGrower<unsigned long, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long, float2, mydouble2,
                HistTreeGrower<unsigned long, float2, mydouble2>>(
                param, data, cfg, obj, verbose.booster);
          } else {
            if (param.method == Exact)
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float2, float2,
                ContinuousTreeGrower<unsigned long long, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
            else
              _builder = new ContinuousGardenBuilder<
                unsigned long long, float2, float2,
                HistTreeGrower<unsigned long long, float2, float2>>(
                param, data, cfg, obj, verbose.booster);
          }
        } else {
          throw "unsupported dimensionality";
        }
        _objective = obj;
      } break;
        //   case SoftMaxOneVsAll: {
        //     auto obj =
        //       new SoftMaxObjective(data, param.labels_count,
        //       param.initial_y);

        //     if (data->max_feature_size + 1 <= sizeof(unsigned char) *
        //     CHAR_BIT) {
        //       if (cfg.double_precision) {
        //         _builder = new ContinuousGardenBuilder<unsigned, float2,
        //         mydouble2>(
        //           param, data, cfg, obj, verbose.booster);
        //       } else {
        //         _builder = new ContinuousGardenBuilder<unsigned, float2,
        //         float2>(
        //           param, data, cfg, obj, verbose.booster);
        //       }
        //     } else if (data->max_feature_size + 1 <=
        //                sizeof(unsigned short) * CHAR_BIT) {
        //       if (cfg.double_precision) {
        //         _builder = new ContinuousGardenBuilder<unsigned, float2,
        //         mydouble2>(
        //           param, data, cfg, obj, verbose.booster);
        //       } else {
        //         _builder = new ContinuousGardenBuilder<unsigned, float2,
        //         float2>(
        //           param, data, cfg, obj, verbose.booster);
        //       }
        //     } else if (data->max_feature_size + 1 <=
        //                sizeof(unsigned int) * CHAR_BIT) {
        //       if (cfg.double_precision) {
        //         _builder =
        //           new ContinuousGardenBuilder<unsigned int, float2,
        //           mydouble2>(
        //             param, data, cfg, obj, verbose.booster);
        //       } else {
        //         _builder =
        //           new ContinuousGardenBuilder<unsigned int, float2, float2>(
        //             param, data, cfg, obj, verbose.booster);
        //       }
        //     } else if (data->max_feature_size + 1 <=
        //                sizeof(unsigned long) * CHAR_BIT) {
        //       if (cfg.double_precision) {
        //         _builder =
        //           new ContinuousGardenBuilder<unsigned long, float2,
        //           mydouble2>(
        //             param, data, cfg, obj, verbose.booster);
        //       } else {
        //         _builder =
        //           new ContinuousGardenBuilder<unsigned long, float2, float2>(
        //             param, data, cfg, obj, verbose.booster);
        //       }
        //     } else if (data->max_feature_size + 1 <=
        //                sizeof(unsigned long long) * CHAR_BIT) {
        //       if (cfg.double_precision) {
        //         _builder = new ContinuousGardenBuilder<unsigned long long,
        //         float2,
        //                                                mydouble2>(
        //           param, data, cfg, obj, verbose.booster);
        //       } else {
        //         _builder =
        //           new ContinuousGardenBuilder<unsigned long long, float2,
        //           float2>(
        //             param, data, cfg, obj, verbose.booster);
        //       }
        //     } else {
        //       throw "unsupported depth";
        //     }
        //     _objective = obj;
        //   } break;
      default:
        throw "Unknown objective function " + param.objective;
    }

    // auto mem_per_rec = _builder->MemoryRequirementsPerRecord();
    size_t total;
    size_t free;

    hipMemGetInfo(&free, &total);

    if (verbose.gpu) {
      printf("Total bytes %ld avaliable %ld \n", total, free);
      //   printf("Memory usage estimation %ld per record %ld in total \n",
      //          mem_per_rec, mem_per_rec * data->rows);
    }

    if (cfg.upload_features) data->TransferToGPU(free * 9 / 10, verbose.gpu);

    _init = true;
  }

  if (grad == NULL) {
    _objective->UpdateGrad();
  } else {
    //          todo: fix
    //          data->grad = std::vector<float>(grad, grad + data->rows);
  }

  for (unsigned short i = 0; i < param.labels_count; ++i) {
    RegTree *tree = new RegTree(param.depth, i);
    _builder->GrowTree(tree, data, i);
    _trees.push_back(tree);
    if (grad == NULL) {
      //   tree->PredictByQuantized(data, data->y_internal);
      //   _builder->PredictByGrownTree(tree, data, data->y_internal);
    }
  }
}

void Garden::UpdateByLastTree(io::DataMatrix *data) {
  if (data->y_internal.size() == 0)
    data->y_internal.resize(data->rows * param.labels_count,
                            _objective->IntoInternal(param.initial_y));
  for (auto it = _trees.end() - param.labels_count; it != _trees.end(); ++it) {
    (*it)->Predict(data, data->y_internal);
  }
}

void Garden::GetY(arboretum::io::DataMatrix *data,
                  std::vector<float> &out) const {
  out.resize(data->y_internal.size());
  _objective->FromInternal(data->y_internal, out);
}

void Garden::Predict(const arboretum::io::DataMatrix *data,
                     std::vector<float> &out) const {
  out.resize(data->rows * param.labels_count);
  thrust::host_vector<float> tmp(data->rows * param.labels_count);

  thrust::fill(tmp.begin(), tmp.end(),
               _objective->IntoInternal(param.initial_y));
  for (size_t i = 0; i < _trees.size(); ++i) {
    _trees[i]->Predict(data, tmp);
  }

  _objective->FromInternal(tmp, out);
}

Garden::~Garden() {
  if (_builder) delete _builder;
  if (_objective) delete _objective;
  for (size_t i = 0; i < _trees.size(); ++i) {
    delete _trees[i];
  }
}

}  // namespace core
}  // namespace arboretum
